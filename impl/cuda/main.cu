#include "hip/hip_runtime.h"

#include <chrono>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <fstream>
#include <iostream>
#include <vector>

using namespace std;

__global__ void cudaFeedForward(double *in, double *weights, double *output,
                                double *activations, int n, int f, int o) {
  // x <-- 0..n
  int ROW = blockIdx.x * blockDim.x + threadIdx.x;
  // y <-- 0..o
  int COL = blockIdx.y * blockDim.y + threadIdx.y;

  if (ROW < n && COL < o) {
    double total = weights[f * o + COL];
    for (int i = 0; i < f; i++) {
      total += in[ROW * n + i] * weights[i * o + COL];
    }
    int index = f * o * COL;
    output[index] = total;

    // Activation
    if (total <= 0.0) {
      total = 0.0;
    }
    activations[index] = total;
  }
}

__global__ void cudaBackPropagateOutput(double *a_j, double *in_o, double *a_o,
                                        double *exp, double *out_error,
                                        double *out_nudge, int n, int f) {
  // x <-- 0..n
  int ROW = blockIdx.x * blockDim.x + threadIdx.x;
  // y <-- 0..f
  int COL = blockIdx.y * blockDim.y + threadIdx.y;

  if (ROW < n && COL < f + 1) {
    double error = 0.0;
    if (a_o[ROW] > 0.0) {
      error = exp[ROW] - a_o[ROW];
    }

    if (COL == f) {
      out_error[ROW] = error;
      out_nudge[(ROW + 1) * (f + 1) - 1] = error;
    } else {
      out_nudge[ROW * (f + 1) + COL] = a_j[f * ROW + COL] * error;
    }
  }
}

__global__ void cudaBackPropagation(double *a_i, double *in_j, double *a_j,
                                    double *w_k, double *err_k,
                                    double *out_nudge, double *out_err_j, int N,
                                    int F, int O, int P) {

	// 0..N
	int n = blockIdx.x * blockDim.x + threadIdx.x;



  // // x <-- 0..n
  // int ROW = blockIdx.x * blockDim.x + threadIdx.x;
  // // y <-- 0..o
  // int COL = blockIdx.y * blockDim.y + threadIdx.y;
  //
  // if (ROW < N && COL < O + 1) {
  //
  //   if (COL == O) {
  //     // bias
  //   } else {
  //     double error = 0.0;
  //     // Derivative of in_j
  //     if (in_j[ROW * O + COL] > 0.0) {
  //       for (int k = 0; k < P; k++) {
  //         error += w_k[COL * P + k] * err_k[ROW * O + k];
  //       }
  //     }
  //
		// 	out_err_j[ROW * O + COL]
  //
  //   }
  // }
}

class CudaTensor {
private:
  int rows;
  int cols;
  double *cuda;

public:
  CudaTensor() : rows(0), cols(0), cuda(nullptr) {}

  CudaTensor(int rows, int cols) : cols(cols), rows(rows) {
    hipMalloc(&this->cuda, this->cols * this->rows * sizeof(double));
  }

  double *getCuda() { return this->cuda; }

  void setValues(double *values) {
    hipMemcpy(this->cuda, values, this->cols * this->rows * sizeof(double),
               hipMemcpyHostToDevice);
  }

  void getValues(double *values) {
    hipMemcpy(values, this->cuda, this->cols * this->rows * sizeof(double),
               hipMemcpyDeviceToHost);
  }

  void free() { hipFree(this->cuda); }
};

class NetworkLayer {
private:
  CudaTensor weights;
  CudaTensor bias;

public:
  int input;
  int output;
  NetworkLayer(int input, int output)
      : input(input), output(output), weights(input, output), bias(output, 1) {}

  void setWeights(double *values) { this->weights.setValues(values); }

  void setBias(double *bias) { this->bias.setValues(bias); }

  void feedForward(double *cuda_in, int observations, double *cuda_out) {
    dim3 threadsPerBlock(this->output, observations);
    dim3 blocksPerGrid(1, 1);

    if (observations * this->output > 512) {
      threadsPerBlock.x = 512;
      threadsPerBlock.y = 512;
      blocksPerGrid.x = ceil(double(this->output) / double(threadsPerBlock.x));
      blocksPerGrid.y = ceil(double(observations) / double(threadsPerBlock.y));
    }

    // cudaFeedForward << <blocksPerGrid, threadsPerBlock >> > (
    //   cuda_in, this->weights.getCuda(), this->bias.getCuda(), cuda_out,
    //   observations, this->input, this->output);
  }

  void free() {
    this->weights.free();
    this->bias.free();
  }
};

class Network {
private:
  int layer_count;
  vector<NetworkLayer> layers;

public:
  int features;
  Network(ifstream stream) {
    this->features = -1;

    string line;

    while (getline(stream, line)) {
      // Line is currently <in> <out>
      int index = line.find(' ');
      int input = stoi(line.substr(0, index));
      if (this->features == -1) {
        this->features = input;
      }
      int output = stoi(line.substr(index + 1));

      // Next line is the bias
      getline(stream, line);
      double bias[output];
      for (int o = 0; o < output - 1; o++) {
        index = line.find(' ');
        bias[o] = stod(line.substr(0, index));
        line = line.substr(index + 1);
      }
      bias[output - 1] = stod(line);

      // Weights
      double weights[input * output];
      for (int i = 0; i < input; i++) {
        getline(stream, line);
        for (int o = 0; o < output - 1; o++) {
          index = line.find(',');
          weights[i * output + o] = stod(line.substr(0, index));
          line = line.substr(index + 1);
        }
        weights[(i + 1) * output - 1] = stod(line);
      }
      getline(stream, line);

      // Now we pushit into a layer
      NetworkLayer layer = NetworkLayer(input, output);
      layer.setWeights(weights);
      layer.setBias(bias);
      layers.push_back(layer);
    }
    stream.close();
    layer_count = layers.size();
  }

  // RELU 1 if positive else negative

  // TODO: convert inputs to be cuda so we can copy to GPU outside of timed
  // sections
  void train(int observations, double *cuda_inputs, double *cuda_expected) {

    double *outputs[layer_count];

    hipMalloc(&outputs[0],
               sizeof(double) * layers.at(0).output * observations);
    layers.at(0).feedForward(cuda_inputs, observations, outputs[0]);

    for (int i = 1; i < layer_count; i++) {
      hipMalloc(&outputs[i],
                 sizeof(double) * layers.at(i).output * observations);
      layers.at(i).feedForward(outputs[i - 1], observations, outputs[i]);
    }

    double *errors[layer_count];

    for (int i = 0; i < layer_count; i++) {
      hipMalloc(&errors[i],
                 sizeof(double) * observations * layers.at(i).output);
    }

    // // TODO MAYBE REBUILD THIS CAUSE I DONT KNOW WHAT I'M DOING
    // runCudaBackPropagateErrorOutput(outputs[layer_count - 1], cuda_expected,
    //                                 errors[layer_count - 1], observations);

    hipFree(cuda_inputs);
    hipFree(cuda_expected);
    for (int i = 0; i < layer_count; i++) {
      hipFree(outputs[i]);
      hipFree(errors[i]);
    }
  }

  ~Network() {
    for (NetworkLayer layer : this->layers) {
      layer.free();
    }
  }
};

int main() {

  Network network = Network(ifstream(getenv("NETWORK")));
  int features = network.features;

  vector<double> data;

  // Read data file
  ifstream stream(getenv("DATA"));
  string line;
  while (getline(stream, line)) {

    int index;
    for (int i = 0; i < network.features; i++) {
      index = line.find(',');
      data.push_back(stod(line.substr(0, index)));
      line = line.substr(index + 1);
    }
    data.push_back(stod(line));
  }
  stream.close();

  vector<vector<int>> bootstraps;
  stream = ifstream(getenv("BOOTSTRAP"));
  while (getline(stream, line)) {
    vector<int> choices;
    int index;
    while ((index = line.find(',')) >= 0) {
      choices.push_back(stoi(line.substr(0, index)));
      line = line.substr(index + 1);
    }
    choices.push_back(stoi(line));

    bootstraps.push_back(choices);
  }

  vector<std::chrono::nanoseconds::rep> times;

  for (vector<int> bootstrap : bootstraps) {
    double train_data[network.features * bootstrap.size()];
    double train_expected[bootstrap.size()];

    for (int i = 0; i < bootstrap.size(); i++) {
      int index = bootstrap.at(i);
      train_expected[i] = data.at((features + 1) * index);
      for (int j = 0; j < features; j++) {
        train_data[i + j] = data.at((features + 1) * index + 1 + j);
      }
    }

    double *cuda_inputs;
    hipMalloc(&cuda_inputs, sizeof(double) * bootstrap.size() * features);
    hipMemcpy(cuda_inputs, train_data,
               sizeof(double) * bootstrap.size() * features,
               hipMemcpyHostToDevice);

    double *cuda_expected;
    hipMalloc(&cuda_expected, sizeof(double) * bootstrap.size());
    hipMemcpy(cuda_expected, train_expected, sizeof(double) * bootstrap.size(),
               hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();

    network.train(bootstrap.size(), cuda_inputs, cuda_expected);

    auto end = std::chrono::high_resolution_clock::now();

    times.push_back(
        std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
            .count());
  }

  ofstream out_file(getenv("OUT_TIMES"));
  if (out_file.is_open()) {
    out_file << "id,time\n";
    for (int i = 0; i < times.size(); ++i) {
      out_file << i << "," << times.at(i) << "\n";
    }

    out_file.close();
  }

  data.clear();
  bootstraps.clear();
}
