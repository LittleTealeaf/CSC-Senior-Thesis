#include "hip/hip_runtime.h"

#include <chrono>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>

using namespace std;

__global__ void cudaMatrixMultiply(double *vec, double *mat, double *res) {}

int main() {

  double vec[] = {0.1, 0.2, 0.3};

  double mat[5 * 3];
  for (int i = 0; i < 5 * 3; i++) {
    mat[i] = 0.1 * i;
  }

  double *cudaVec = 0;
  double *cudaMat = 0;
  double *cudaRes = 0;

  hipMalloc(&cudaVec, sizeof(vec));
  hipMalloc(&cudaMat, sizeof(mat));
  hipMalloc(&cudaRes, sizeof(double) * 3 * 5 * 3);

  hipMemcpy(cudaVec, vec, sizeof(vec), hipMemcpyHostToDevice);
  hipMemcpy(cudaMat, mat, sizeof(mat), hipMemcpyHostToDevice);

  hipFree(&cudaVec);
  hipFree(&cudaMat);
  hipFree(&cudaRes);

  return 0;
}
