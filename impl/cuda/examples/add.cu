#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include <chrono>
#include <iostream>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

using namespace std;

__global__ void vectorAdd(int* a, int* b, int* c) {
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

int main() {

  for (int i = 0; i < 100; i++) {

    int a[] = { 1, 2, 3 };
    int b[] = { 4, 5, 6 };

    int c[sizeof(a) / sizeof(int)] = { 0 };

    int* cudaA = 0;
    int* cudaB = 0;
    int* cudaC = 0;

    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));

    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();

    vectorAdd << <1, sizeof(a) / sizeof(int) >> > (cudaA, cudaB, cudaC);

    auto finish = std::chrono::high_resolution_clock::now();

    hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

    hipFree(cudaA);
    hipFree(cudaB);
    hipFree(cudaC);

    cout << std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start)
      .count()
      << endl;
  }

  return 0;
}
